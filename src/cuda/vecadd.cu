#include "hip/hip_runtime.h"
#include "vecadd.cuh"

__global__ void i_vector_add(float *out, float *a, float *b, int n) {
  int index = threadIdx.x;
  int stride = blockDim.x;

  for (int i = index; i < n; i += stride) {
    out[i] = a[i] + b[i];
  }
}

namespace cu {
void vector_add(float *out, float *a, float *b, int n) {
  i_vector_add<<<1,256>>>(out, a, b, n);
}
} // namespace cu